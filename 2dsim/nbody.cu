#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <pthread.h>
#include <unistd.h>
#include <sys/mman.h>
#include <fcntl.h>
#include <math.h>
#include <fstream>
#include <dirent.h>
#include "json.hpp"

using json = nlohmann::json;

#define DEBUG true
#define N 1000     // max number of bodies
#define G 6.67430e-11 // Gravitational constant in m³ kg⁻¹ s⁻²
#define DT 1  // Time step in seconds

struct Body {
    double3 position;
    double3 velocity;
    float3 color;
    double mass;
    double radius;
    char name[50];
    bool active;
    int parentIndex; // -1 if no parent
};

Body* bodies;
Body* d_bodies;
int activeBodyCount = 0;

__global__ void updateBodies(Body* bodies, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n && bodies[i].active) {
        double3 force = {0.0, 0.0, 0.0};
        for (int j = 0; j < n; j++) {
            if (i != j && bodies[j].active) {
                double3 r;
                r.x = bodies[j].position.x - bodies[i].position.x;
                r.y = bodies[j].position.y - bodies[i].position.y;
                r.z = bodies[j].position.z - bodies[i].position.z;

                double distSqr = r.x * r.x + r.y * r.y + r.z * r.z;
                double F = G * bodies[i].mass * bodies[j].mass *
                    rsqrt(distSqr * distSqr * distSqr);
                force.x += F * r.x;
                force.y += F * r.y;
                force.z += F * r.z;
            }
        }

        bodies[i].velocity.x += DT * force.x / bodies[i].mass;
        bodies[i].velocity.y += DT * force.y / bodies[i].mass;
        bodies[i].velocity.z += DT * force.z / bodies[i].mass;

        bodies[i].position.x += DT * bodies[i].velocity.x;
        bodies[i].position.y += DT * bodies[i].velocity.y;
        bodies[i].position.z += DT * bodies[i].velocity.z;
    }
}

void* simulationThread(void* arg) {
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;

    while (true) {
        updateBodies<<<numBlocks, blockSize>>>(d_bodies, N);
        hipDeviceSynchronize();
        hipMemcpy(bodies, d_bodies, N * sizeof(Body), hipMemcpyDeviceToHost);
        usleep(1000);
    }
}

void addBody(const char* name,
             double3 position,
             double3 velocity,
             double mass,
             double radius,
             float3 color,
             int parentIndex = -1) {

    if (activeBodyCount >= N) {
        printf("Warning: Maximum number of bodies reached. Ignoring new body.\n");
        return;
    }

    int i = activeBodyCount;
    strcpy(bodies[i].name, name);
    bodies[i].position = position;
    bodies[i].velocity = velocity;
    bodies[i].mass = mass;
    bodies[i].radius = radius;
    bodies[i].color = color;
    bodies[i].active = true;
    bodies[i].parentIndex = parentIndex;
    ++activeBodyCount;
}

int get_body_by_name(const char* name) {
    for (int i = 0; i < N; i++) {
        if (bodies[i].active && strcmp(bodies[i].name, name) == 0) {
            return i;
        }
    }
    return -1;
}

void loadBodiesFromJson(
    const std::string& filename,
    double3 position_offset = {0.0, 0.0, 0.0},
    double3 velocity_offset = {0.0, 0.0, 0.0}) {

    std::ifstream file(filename);
    json j;
    file >> j;

    double3 parPos = position_offset;
    double3 parVel = velocity_offset;
    int parIdx = -1;

    if (j.contains("parent")) {
        const auto& par = j["parent"];
        const char* parentName = par["name"].get<std::string>().c_str();
        // check to make sure parentName is not already in bodies. if so,
        // we use the parent's position and velocity as the offset and we
        // do not touch the parent
        parIdx = get_body_by_name(parentName);
        if (parIdx != -1) {
            parPos = bodies[parIdx].position;
            parVel = bodies[parIdx].velocity;
        }
        else {
            // update parent position and velocity
            double3 relPos = make_double3(par["position"][0].get<double>(),
                                          par["position"][1].get<double>(),
                                          par["position"][2].get<double>());
            double3 relVel = make_double3(par["velocity"][0].get<double>(),
                                          par["velocity"][1].get<double>(),
                                          par["velocity"][2].get<double>());
            addBody(parentName,
                    make_double3(parPos.x + relPos.x,
                                 parPos.y + relPos.y,
                                 parPos.z + relPos.z),
                    make_double3(parVel.x + relVel.x,
                                 parVel.y + relVel.y,
                                 parVel.z + relVel.z),
                    par["mass"].get<double>(),
                    par["radius"].get<double>(),
                    make_float3(par["color"][0].get<float>(),
                                par["color"][1].get<float>(),
                                par["color"][2].get<float>()));
            parIdx = activeBodyCount - 1;
        }
    }

    for (const auto& body : j["children"]) {
        const char* childName = body["name"].get<std::string>().c_str();
        int childIdx = get_body_by_name(childName);
        // warn if child already exists
        if (childIdx != -1) {
            printf("Warning: Body with name %s already exists. Ignoring new body.\n", childName);
            continue;
        }
        addBody(childName,
                make_double3(parPos.x + body["position"][0].get<double>(),
                             parPos.y + body["position"][1].get<double>(),
                             parPos.z + body["position"][2].get<double>()),
                make_double3(parVel.x + body["velocity"][0].get<double>(),
                             parVel.y + body["velocity"][1].get<double>(),
                             parVel.z + body["velocity"][2].get<double>()),
                body["mass"].get<double>(),
                body["radius"].get<double>(),
                make_float3(body["color"][0].get<float>(),
                            body["color"][1].get<float>(),
                            body["color"][2].get<float>()),
                parIdx);
    }

    // Update device memory
    hipMemcpy(d_bodies, bodies, N * sizeof(Body), hipMemcpyHostToDevice);
}

void loadBodiesFromJsonDir(const std::string& dirPath) {
    DIR* dir;
    struct dirent* ent;
    if ((dir = opendir(dirPath.c_str())) != NULL) {
        while ((ent = readdir(dir)) != NULL) {
            std::string filename = ent->d_name;
            if (filename.find(".json") != std::string::npos) {
                std::string filepath = dirPath + "/" + filename;
                loadBodiesFromJson(filepath);
            }
        }
        closedir(dir);
    } else {
        perror("opendir");
    }
}

int main() {
    // Use shm_open to create a shared memory segment
    int shm_fd = shm_open("/bodies", O_CREAT | O_RDWR, 0666);
    if (shm_fd == -1) {
        perror("shm_open");
        return 1;
    }
    if (ftruncate(shm_fd, N * sizeof(Body)) == -1) {
        perror("ftruncate");
        return 1;
    }
    bodies = (Body*)mmap(NULL, N * sizeof(Body), PROT_READ | PROT_WRITE, MAP_SHARED, shm_fd, 0);
    if (bodies == MAP_FAILED) {
        perror("mmap");
        return 1;
    }

    for (int i = 0; i < N; i++) {
        bodies[i].active = false;
    }

    hipMalloc(&d_bodies, N * sizeof(Body));
    loadBodiesFromJsonDir("./solar_system");
    //loadBodiesFromJson("./solar_system/main_planets.json");
    //loadBodiesFromJson("./solar_system/jupiter.json");
    hipMemcpy(d_bodies, bodies, N * sizeof(Body), hipMemcpyHostToDevice);

    pthread_t thread;
    pthread_create(&thread, NULL, simulationThread, NULL);

    printf("[simulation]: running with %d bodies\n", activeBodyCount);

    // Keep the main thread alive to allow queries
    while (true) {
        if (DEBUG) {
            for (int i = 0; i < N; i++) {
                if (!bodies[i].active) {
                    continue;
                }
                // make sure to show in scientific notation
                printf("Name: %s\n", bodies[i].name);
                printf("\tPosition: (%f, %f, %f)\n", bodies[i].position.x, bodies[i].position.y, bodies[i].position.z);
                printf("\tVelocity: (%f, %f, %f)\n", bodies[i].velocity.x, bodies[i].velocity.y, bodies[i].velocity.z);
                printf("\tMass: %f kg\n", bodies[i].mass);
                printf("\tRadius: %f m\n", bodies[i].radius);
                printf("\n");                
            }
        }
        sleep(10);       
    }

    return 0;
}